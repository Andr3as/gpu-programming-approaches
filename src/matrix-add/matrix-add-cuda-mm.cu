#include "hip/hip_runtime.h"
#include "matrix-add-util.h"

#include "../cuda-util.h"


__global__ void matrixAdd(const double *__restrict__ a, const double *__restrict__ b, double *__restrict__ c, size_t nx, size_t ny) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t i1 = blockIdx.y * blockDim.y + threadIdx.y;

    if (i0 < nx && i1 < ny) {
        c[i0 + i1 * nx] = a[i0 + i1 * nx] + b[i0 + i1 * nx];
    }
}


int main(int argc, char *argv[]) {
    size_t nx, ny, nItWarmUp, nIt;
    parseCLA_2d(argc, argv, nx, ny, nItWarmUp, nIt);

    double *a;
    checkCudaError(hipMallocManaged((void **)&a, sizeof(double) * nx * ny));
    double *b;
    checkCudaError(hipMallocManaged((void **)&b, sizeof(double) * nx * ny));
    double *c;
    checkCudaError(hipMallocManaged((void **)&c, sizeof(double) * nx * ny));

    // init
    initMatrixAdd(a, b, c, nx, ny);

    checkCudaError(hipMemPrefetchAsync(a, sizeof(double) * nx * ny, 0));
    checkCudaError(hipMemPrefetchAsync(b, sizeof(double) * nx * ny, 0));
    checkCudaError(hipMemPrefetchAsync(c, sizeof(double) * nx * ny, 0));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        matrixAdd<<<dim3(ceilingDivide(nx, 16), ceilingDivide(ny, 16)), dim3(16, 16)>>>(a, b, c, nx, ny);
        std::swap(c, a);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        matrixAdd<<<dim3(ceilingDivide(nx, 16), ceilingDivide(ny, 16)), dim3(16, 16)>>>(a, b, c, nx, ny);
        std::swap(c, a);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats(end - start, nIt, nx * ny, sizeof(double) + sizeof(double) + sizeof(double), 1);

    checkCudaError(hipMemPrefetchAsync(a, sizeof(double) * nx * ny, hipCpuDeviceId));
    checkCudaError(hipMemPrefetchAsync(b, sizeof(double) * nx * ny, hipCpuDeviceId));
    checkCudaError(hipMemPrefetchAsync(c, sizeof(double) * nx * ny, hipCpuDeviceId));

    // check solution
    checkSolutionMatrixAdd(a, b, c, nx, ny, nIt + nItWarmUp);

    checkCudaError(hipFree(a));
    checkCudaError(hipFree(b));
    checkCudaError(hipFree(c));

    return 0;
}
