#include "hip/hip_runtime.h"
#include "matrix-add-util.h"

#include "../cuda-util.h"


__global__ void matrixAdd(const double *__restrict__ a, const double *__restrict__ b, double *__restrict__ c, size_t nx, size_t ny) {
    const size_t i0 = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t i1 = blockIdx.y * blockDim.y + threadIdx.y;

    if (i0 < nx && i1 < ny) {
        c[i0 + i1 * nx] = a[i0 + i1 * nx] + b[i0 + i1 * nx];
    }
}


int main(int argc, char *argv[]) {
    size_t nx, ny, nItWarmUp, nIt;
    parseCLA_2d(argc, argv, nx, ny, nItWarmUp, nIt);

    double *a;
    checkCudaError(hipHostMalloc((void **)&a, sizeof(double) * nx * ny));
    double *b;
    checkCudaError(hipHostMalloc((void **)&b, sizeof(double) * nx * ny));
    double *c;
    checkCudaError(hipHostMalloc((void **)&c, sizeof(double) * nx * ny));

    double *d_a;
    checkCudaError(hipMalloc((void **)&d_a, sizeof(double) * nx * ny));
    double *d_b;
    checkCudaError(hipMalloc((void **)&d_b, sizeof(double) * nx * ny));
    double *d_c;
    checkCudaError(hipMalloc((void **)&d_c, sizeof(double) * nx * ny));

    // init
    initMatrixAdd(a, b, c, nx, ny);

    checkCudaError(hipMemcpy(d_a, a, sizeof(double) * nx * ny, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_b, b, sizeof(double) * nx * ny, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_c, c, sizeof(double) * nx * ny, hipMemcpyHostToDevice));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        matrixAdd<<<dim3(ceilingDivide(nx, 16), ceilingDivide(ny, 16)), dim3(16, 16)>>>(d_a, d_b, d_c, nx, ny);
        std::swap(d_c, d_a);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        matrixAdd<<<dim3(ceilingDivide(nx, 16), ceilingDivide(ny, 16)), dim3(16, 16)>>>(d_a, d_b, d_c, nx, ny);
        std::swap(d_c, d_a);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats(end - start, nIt, nx * ny, sizeof(double) + sizeof(double) + sizeof(double), 1);

    checkCudaError(hipMemcpy(a, d_a, sizeof(double) * nx * ny, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(b, d_b, sizeof(double) * nx * ny, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(c, d_c, sizeof(double) * nx * ny, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionMatrixAdd(a, b, c, nx, ny, nIt + nItWarmUp);

    checkCudaError(hipFree(d_a));
    checkCudaError(hipFree(d_b));
    checkCudaError(hipFree(d_c));

    checkCudaError(hipHostFree(a));
    checkCudaError(hipHostFree(b));
    checkCudaError(hipHostFree(c));

    return 0;
}
